#include "hip/hip_runtime.h"
#include "image.hh"

float saturatef(float x)
{
    return min(1.0, max(x, 0.0));
}

Image::Image(size_t x_dim, size_t y_dim, bool hostOnly) :
    shape(x_dim, y_dim), deviceData(nullptr), hostData(nullptr),
    deviceAllocated(false), hostAllocated(false), hostOnly(hostOnly)
{ }

Image::Image(Shape shape, bool hostOnly) :
    Image(shape.x, shape.y, hostOnly)
{ }

void Image::allocateDeviceMemory() {
    if (!deviceAllocated) {
        hipError_t ok;
        uint * deviceMemory = nullptr;

        ok = hipMalloc(&deviceMemory, shape.x * shape.y * sizeof(uint));
        checkCudaErrors(ok);
        deviceData = std::shared_ptr<uint> (deviceMemory, [&](uint* ptr){ hipFree(ptr); });
        deviceAllocated = true;
    }
}

void Image::allocateHostMemory() {
    if (!hostAllocated) {
        hostData = std::shared_ptr<uint> (new uint[shape.x*shape.y], [&](uint* ptr){ delete[] ptr; });
        hostAllocated = true;
    }
}

bool Image::loadPNG(std::string filename) {
    std::vector<unsigned char> png; 
    unsigned w, h;

    // get h and w to allocate memory
    unsigned error = lodepng::decode(png, w, h, filename);

    if (error) {
        std::cout << "Error reading png: " << lodepng_error_text(error) << std::endl;
        return false;
    }

    maybeAllocateMemory(Shape((int)w,(int)h));

    uint r, g, b, a;

    // copy into host memory.
    for (int i = 0 ; i < (png.size()/4); i ++) {
        r = png[i*4]; 
        g = png[i*4+1];
        b = png[i*4+2];
        a = png[i*4+3];
        hostData.get()[i] = (uint(a)<<24) | (uint(b)<<16) | (uint(g)<<8) | uint(r);
    }

    // copy into device memory (should be constant...)
    copyHostToDevice();

    return true;
}

bool Image::savePNG(std::string filename, bool doFlip, bool doMirror){
    std::vector<unsigned char> png;
    
    if (!hostAllocated) { 
        std::cout << "[ERROR] no data to save...\n"; 
        return false;
    }

    unsigned char r,g,b,a;
    for (int i = 0; i < size(); i ++) {
        //mask and shift our colors back.
        uint color = hostData.get()[i];
        a = (color & 0xFF000000) >> 24;
        b = (color & 0x00FF0000) >> 16;
        g = (color & 0x0000FF00) >> 8;
        r = (color & 0x000000FF);

        if (doFlip) {
            png.push_back(a);
            png.push_back(b);
            png.push_back(g);
            png.push_back(r);
        } else {
            png.push_back(r);
            png.push_back(g);
            png.push_back(b);
            png.push_back(a);
        }
    } 
    if (doFlip) {
        std::reverse(png.begin(), png.end());
    }
    if (doMirror) {
        
    }

    unsigned error = lodepng::encode(filename, png, shape.x, shape.y);

    if (error) {
        std::cout << "[ERROR] Unable to save png: " << lodepng_error_text(error) << std::endl;
        return false;
    } 
    return true;
}

void Image::allocateMemory() {

    allocateHostMemory();
    
    if (!hostOnly) {
        allocateDeviceMemory();
    }
}

void Image::maybeAllocateMemory(Shape shape) {
    if (!deviceAllocated && !hostAllocated) {
        this->shape = shape;
        allocateMemory();
    } 
}

void Image::copyHostToDevice() {
    if (deviceAllocated && hostAllocated) {
        hipError_t ok;
        ok = hipMemcpy(deviceData.get(), hostData.get(), shape.x * shape.y * sizeof(uint), hipMemcpyHostToDevice);
		checkCudaErrors(ok);
    } else {
        printf("Failed to copy from host to device... nothing initialized\n");
    }
}

void Image::copyDeviceToHost() {
    if (deviceAllocated && hostAllocated) {
        hipError_t ok;
        ok = hipMemcpy(
            hostData.get(), 
            deviceData.get(), 
            shape.x * shape.y * sizeof(uint), 
            hipMemcpyDeviceToHost
        );

        checkCudaErrors(ok);

    } else {
        printf("Failed to copy from device to host... nothing initialized\n");
    }
}

uint& Image::operator[](const int index) {
	return hostData.get()[index];
}

const uint& Image::operator[](const int index) const {
	return hostData.get()[index];
}

