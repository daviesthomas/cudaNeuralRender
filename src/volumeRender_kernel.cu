#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include "neuralNetwork.hh"
#include "layers/denseLayer.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


typedef unsigned int  uint;
typedef unsigned char uchar;

typedef struct
{
    float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};


// intersect ray with a box
__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}


// intersect ray with a box
__device__
int intersectSphere(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{

    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

    *tnear = largest_tmin;
    *tfar = smallest_tmax;

    return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uint rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}

// given a point return distance to surface
__device__ float distanceToSurface(float3 pos)
{
	const float radius = 0.5f;
    return length(pos)-radius;
}

//given a point on surface, calc & retrun normal.
__device__ float3 fragNormal(float3 p)
{
    float3 n;
    float3 a,b;
    const float EPSILON = 0.00001;

    a.x = p.x + EPSILON;
    a.y = p.y;
    a.z = p.z;

    b.x = p.x - EPSILON;
    b.y = p.y;
    b.z = p.z;

    n.x = distanceToSurface(a) - distanceToSurface(b);

    a.x = p.x;
    a.y = p.y + EPSILON;
    b.x = p.x;
    b.y = p.y - EPSILON;

    n.y = distanceToSurface(a) - distanceToSurface(b);

    a.y = p.y;
    a.z = p.z + EPSILON;
    b.y = p.y;
    b.z = p.z - EPSILON;

    n.z = distanceToSurface(a) - distanceToSurface(b);


    return normalize(n);
}

__global__ void relu(float* Z, float* A,
    int Z_x_dim, int Z_y_dim) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < Z_x_dim * Z_y_dim) {
        A[index] = fmaxf(Z[index], 0);
    }
}

__global__ void denseForward(
    float* W, 
    float* A,
    float* Z, 
    float* b,
    int W_x_dim, int W_y_dim,
    int A_x_dim, int A_y_dim, int activation) 
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int Z_x_dim = A_x_dim;
    int Z_y_dim = W_y_dim;

    float Z_value = 0;

    if (row < Z_y_dim && col < Z_x_dim) {
        for (int i = 0; i < W_x_dim; i++) {
            Z_value += W[row * W_x_dim + i] * A[i * A_x_dim + col];
        }
        Z[row * Z_x_dim + col] = Z_value + b[row];
    }
}

__device__ 
float forwardInfer(
    float* weights, 
    float* biases, 
    float* dims, 
    float* A,
    float* Z,
    int numLayers
) 
{    
    // position in mem for each layers weights and biases.
    int wPos = 0;
    int bPos = 0;
    
    for (int i = 0; i < numLayers; i ++) {
        int M = int(dims[i*3]);
        int N = int(dims[i*3+1]);
        int K = int(dims[i*3+2]);

        // just for reference...
        //int Wx = M;
        //int Wy = K;
        //int Ax = N;
        //int Ay = K;
        //int bx = Ax;
        //int by = Wy;
        //int Zx = Ax;
        //int Zy = Wy;

        dim3 blockSize(4, 4);
        dim3 threadsPerBlock(	(M + blockSize.x - 1) / blockSize.x,
                                (K + blockSize.y - 1) / blockSize.y);

        //Dynamic parallelism. 
        denseForward<<<blockSize,threadsPerBlock>>>(
            weights + wPos, 
            A, 
            Z, 
            biases + bPos,
            M,
            K,
            N,
            K,
            0
        );

        hipDeviceSynchronize();

        if (i < (numLayers - 1)){
            relu<<<1,K*N>>>(Z, A, N, K);
            hipDeviceSynchronize();
            // increment pointer to next layer of weights
            wPos += M*K;
            bPos += N*K;
        }
    }
    float Y = tanh(Z[0]);

    return Y;
}

//simple kernel for testing forward inference.
__global__ void 
inferTest(
    float* weights, 
    float* biases, 
    float* dims,
    float* A,
    float* Z,
    int numLayers
) {
    int id = 32;

    A[id+0] = static_cast<float>(-0.245570);
    A[id+1] = static_cast<float>(-0.429746);
    A[id+2] = static_cast<float>( -0.245570);
    float tstep = forwardInfer(weights, biases, dims, A+id, Z+id, numLayers);
    printf("tstep: %f \n", tstep);
}


__global__ void
d_render(
    uint *d_output, 
    uint imageW, 
    uint imageH, 
    float* weights, 
    float* biases, 
    float* dims,
    float* A,
    float* Z,
    int numLayers)
{
    const int maxSteps = 60;
    
    const float EPSILON = 0.00001;
    const float3 boxMin = make_float3(-0.5f, -0.5f, -0.5f);
    const float3 boxMax = make_float3(0.5f, 0.5f, 0.5f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;
    int id = x*imageW + y;

    if ((x >= imageW) || (y >= imageH)) return;

    float u = (x / (float) imageW)*2.0f-1.0f;
    float v = (y / (float) imageH)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, -2.0f));
    eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    // no need to march if ray never hits sphere!
    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    float tstep;
    float t = tnear;
    // start ray at edge of bounds
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step;

    // march the ray!
    for (int i=0; i<maxSteps; i++)
    {
        // get dist to surface
        A[id*32+0] = static_cast<float>(pos.x);
        A[id*32+1] = static_cast<float>(pos.y);
        A[id*32+2] = static_cast<float>(pos.z);

        //tstep = -forwardInfer(weights, biases, dims, A + id*32, Z + id*32, numLayers);
        tstep = distanceToSurface(pos);

        // if close enough, we're done!
        if (tstep < EPSILON) break;
        // step along ray
        step = eyeRay.d*tstep;
        t += tstep;
        // if past bounding box, we're done!
        if (t > tfar) break;
        // step ray for next iter
        pos += step;
    }

    float4 col; ;
    if (tstep < EPSILON) {
        // set color based on normals! (we'll later use matcap to look up in iamge...)

        pos += eyeRay.d * tstep;
        float3 normal = fragNormal(pos);
        col.x = normal.x;
        col.y = normal.y;
        col.z = normal.z;
        col.w = 1.0;

    } else {
        // either left the box OR reached max steps.
        col = make_float4(0.2f);
    }

    // write output color
    d_output[y*imageW + x] = rgbaFloatToInt(col);
}

extern "C"
void render_kernel(
    dim3 gridSize, 
    dim3 blockSize, 
    uint *d_output, 
    uint imageW, 
    uint imageH,
    float* weights, 
    float* biases, 
    float* dims,
    float* A,
    float* Z,
    int numLayers
)
{

    d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, weights, biases, dims, A, Z, numLayers);
}

extern "C"
void sdf_kernel(
    float* weights, 
    float* biases, 
    float* dims,
    float* A,
    float* Z,
    int numLayers
)
{
    inferTest<<<1,1>>>(weights, biases, dims, A, Z, numLayers);
}

extern "C"
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}

#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
